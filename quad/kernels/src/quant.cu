#include "hip/hip_runtime.h"
#include <quant.h>

template <typename T>
__device__ __half int_to_half(T value)
{
    return __int2half_rn(static_cast<int>(value));
}

template <typename T>
__device__ float int_to_float(T value)
{
    return __int2float_rn(static_cast<float>(value));
}

template <typename T>
__device__ __half float_to_half(T value)
{
    return __float2half(static_cast<float>(value));
}

template <typename T>
__device__ __hip_bfloat16 float_to_bf16(T value)
{
    return __float2bfloat16(static_cast<float>(value));
}

__global__ void sym_quantize_f16_i4_kernel(
    const half *__restrict__ x,
    const half *__restrict__ scale,
    uint32_t rows,
    uint32_t colsSrc,
    uint32_t colsDst,
    uint32_t stride_row,
    Int4Storage *__restrict__ q)
{
    uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t colDst = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= rows || colDst * kElementsPerVector >= colsSrc)
    {
        return;
    }
    Int4Storage storage;
    memset(&storage, 0, sizeof(storage));
    uint32_t id = colDst * kElementsPerVector + row * stride_row;
#pragma unroll
    for (int i = 0; i < kElementsPerVector; ++i)
    {
        bool safe = (colDst * kElementsPerVector + i) < colsSrc;
        if (safe)
        {
            half data = __hdiv(x[id + i], scale[row]);

            int qval = clamp(__half2int_rn(data), qmin, qmax);
            Int4Subbyte{reinterpret_cast<cutlass::int4b_t *>(&storage), i}.set(
                qval);
        }
    }

    q[colDst + row * colsDst] = storage;
}

void sym_quant_fp16_i4_host(
    const half *x,
    const half *scale,
    uint32_t rows,
    uint32_t colsSrc,
    uint32_t colsDst,
    uint32_t stride_row,
    Int4Storage *q)
{

    dim3 block{std::min<uint32_t>(colsDst, 32), std::min<uint32_t>(rows, 16)};
    dim3 grid{cdiv(colsDst, block.x), cdiv(rows, block.y)};
    sym_quantize_f16_i4_kernel<<<grid, block>>>(x, scale, rows, colsSrc, colsDst, stride_row, q);
}

__global__ void sym_quantize_f16_i8_kernel(
    const half *__restrict__ x,
    const half *__restrict__ scale,
    uint32_t rows,
    uint32_t cols,
    uint32_t stride_row,
    int8_t *__restrict__ q)
{
    uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t colDst = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= rows || colDst >= cols)
    {
        return;
    }
    int8_t storage;
    memset(&storage, 0, sizeof(storage));
    uint32_t id = colDst + row * stride_row;
    bool safe = colDst < cols;
    if (safe)
    {
        half data = __hdiv(x[id], scale[row]);
        storage = static_cast<int8_t>(clamp(__half2int_rn(data), qmin_int8, qmax_int8));
    }
    q[colDst + row * cols] = storage;
}

void sym_quant_fp16_i8_host(
    const half *x,
    const half *scale,
    uint32_t rows,
    uint32_t cols,
    uint32_t stride_row,
    int8_t *q)
{

    dim3 block{std::min<uint32_t>(cols, 32), std::min<uint32_t>(rows, 16)};
    dim3 grid{cdiv(cols, block.x), cdiv(rows, block.y)};
    sym_quantize_f16_i8_kernel<<<grid, block>>>(x, scale, rows, cols, stride_row, q);
}

__global__ void sym_dequantize_i32_f16_kernel(
    const int32_t *__restrict__ q,
    const half *__restrict__ scale_row,
    const half *__restrict__ scale_col,
    uint32_t rows, uint32_t cols,
    half *__restrict__ x)
{
    uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t col = threadIdx.x + blockIdx.x * blockDim.x;

    if (col >= cols || row >= rows)
    {
        return;
    }

    float xElement = int_to_float(q[col + row * cols]);
    float scale_r = __half2float(scale_row[row]);
    float scale_c = __half2float(scale_col[col]);
    x[col + row * cols] = float_to_half(scale_r * scale_c * xElement);
}

void sym_dequant_host(const int32_t *q,
                      const half *scale_row,
                      const half *scale_col,
                      uint32_t rows,
                      uint32_t cols,
                      half *x)
{
    dim3 block{std::min<uint32_t>(cols, 16), std::min<uint32_t>(rows, 16)};
    dim3 grid{cdiv(cols, block.x), cdiv(rows, block.y)};
    sym_dequantize_i32_f16_kernel<<<grid, block>>>(
        q,
        scale_row, scale_col,
        rows, cols, x);
}

__global__ void sym_dequantize_i4_fp16_kernel(
    const Int4Storage *__restrict__ q,
    const half *__restrict__ scale_row,
    uint32_t rows, uint32_t colsSrc, uint32_t colsDst,
    half *__restrict__ x)
{
    uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= rows || col >= colsSrc)
    {
        return;
    }
    Int4Storage storage = q[col + row * colsSrc];
    uint32_t id = col * kElementsPerVector + row * colsDst;
#pragma unroll
    for (int i = 0; i < kElementsPerVector; ++i)
    {
        bool safe = (col * kElementsPerVector + i) < colsDst;
        if (safe)
        {
            int q_val = cutlass::int4b_t::xint_t(
                Int4Subbyte{reinterpret_cast<cutlass::int4b_t *>(&storage), i}.get());
            float scale_r = __half2float(scale_row[row]);
            x[id + i] = float_to_half(scale_r * q_val);
        }
    }
}

void sym_dequant_fp16_weight_host(const Int4Storage *q,
                             const half *scale_row,
                             uint32_t rows,
                             uint32_t colsSrc,
                             uint32_t colsDst,
                             half *x)
{
    dim3 block{std::min<uint32_t>(colsSrc, 16), std::min<uint32_t>(rows, 16)};
    dim3 grid{cdiv(colsSrc, block.x), cdiv(rows, block.y)};
    sym_dequantize_i4_fp16_kernel<<<grid, block>>>(
        q,
        scale_row,
        rows, colsSrc, colsDst, x);
}

__global__ void sym_dequantize_i4_bf16_kernel(
    const Int4Storage *__restrict__ q,
    const __hip_bfloat16 *__restrict__ scale_row,
    uint32_t rows, uint32_t colsSrc, uint32_t colsDst,
    __hip_bfloat16 *__restrict__ x)
{
    uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= rows || col >= colsSrc)
    {
        return;
    }
    Int4Storage storage = q[col + row * colsSrc];
    uint32_t id = col * kElementsPerVector + row * colsDst;
#pragma unroll
    for (int i = 0; i < kElementsPerVector; ++i)
    {
        bool safe = (col * kElementsPerVector + i) < colsDst;
        if (safe)
        {
            int q_val = cutlass::int4b_t::xint_t(
                Int4Subbyte{reinterpret_cast<cutlass::int4b_t *>(&storage), i}.get());
            float scale_r = __bfloat162float(scale_row[row]);
            x[id + i] = float_to_bf16(scale_r * q_val);
        }
    }
}

void sym_dequant_bf16_weight_host(const Int4Storage *q,
                             const __hip_bfloat16 *scale_row,
                             uint32_t rows,
                             uint32_t colsSrc,
                             uint32_t colsDst,
                             __hip_bfloat16 *x)
{
    dim3 block{std::min<uint32_t>(colsSrc, 16), std::min<uint32_t>(rows, 16)};
    dim3 grid{cdiv(colsSrc, block.x), cdiv(rows, block.y)};
    sym_dequantize_i4_bf16_kernel<<<grid, block>>>(
        q,
        scale_row,
        rows, colsSrc, colsDst, x);
}
